#include "hip/hip_runtime.h"
#include <>
#include "HistogramKernel.h"

#define cudaCheck(stmt)                                                        \
    do {                                                                    \
        hipError_t err = stmt;                                                \
        if (err != hipSuccess) {                                            \
            cudaLog(ERROR, "Failed to run stmt ", #stmt);                    \
            cudaLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err));    \
            return -1;                                                        \
            }                                                                \
        } while (0)                                                            \


__global__ void histogramKernel(int *vec, int n, unsigned int *d_out) {
    // Taking care of the following: Privatization, using shared memory and using atomic operations, num bins mapping
    // Splitting into sections and determining the number of the sections for each thread
    // and howe much the thread will take

    // Block size is 256 thread, so the elements_per_threads will be 50 in each thread
    // We can make use of the caolesced memories without nothing changed
    __shared__ unsigned int s_bins[NUM_BINS];

    int block_start_index = blockIdx.x * blockDim.x * ELEMENTS_PER_THREAD;
    int index = threadIdx.x + block_start_index;

    // Calculating hist in privatization shared memory
    int k = 0;
    while (index < n && k < ELEMENTS_PER_THREAD) {
        int count = atomicAdd(&s_bins[vec[index]], 1);
        printf("blockIndex %d thread %d cur_index %d vectorNumber %d Count %d blockStartIndex %d\n", blockIdx.x, threadIdx.x,
               index,
               vec[index], count, block_start_index);
        index += ELEMENTS_PER_THREAD;
        k++;
        __syncthreads();

    }

    // Add to the global memory
    if (threadIdx.x == 0)
        for (int j = 0; j < NUM_BINS; j++)
            atomicAdd(&d_out[j], s_bins[j]);

}


__global__ void textHistogramKernel(char *vec, int n, unsigned int *out, int index_jump) {
    // Taking care of the following: Privatization, using shared memory and using atomic operations, num bins mapping
    // Splitting into sections and determining the number of the sections for each thread
    // and howe much the thread will take

    // Block size is 256 thread, so the index_jump will be 50 in each thread
    // We can make use of the caolesced memories without nothing changed
    __shared__ unsigned int s_bins[TEXT_NUM_BINS];
    int i_thread = threadIdx.x + blockIdx.x * blockDim.x;

    // Initializing the bin counters
    int i = i_thread;
    if (i < TEXT_NUM_BINS)
        s_bins[i] = 0;
    __syncthreads();

    // Calculating hist in privatization shared memory
    while (i < n) {
        // Atomic add
        atomicAdd(&s_bins[int(vec[i])], 1);
        i += index_jump;

        __syncthreads();
    }

    // Add to the global memory, as the block has 256 threads (more than the number of bins)
    if (i_thread < TEXT_NUM_BINS) {
        atomicAdd(&out[i_thread], s_bins[i_thread]);
    }
}

void HistogramKernel::runHistogram(int *vec, int n, unsigned int *out) {
    histogramKernel << < this->gridSize, this->blockSize >> > (vec, n, out);
}

void HistogramKernel::runTextHistogram(char *d_in, int n, unsigned int *out) {
    textHistogramKernel << < this->gridSize, this->blockSize >> > (d_in, n, out, ELEMENTS_PER_THREAD);
}
#include "hip/hip_runtime.h"
#include <>
#include "HistogramKernel.h"

#define cudaCheck(stmt)                                                        \
    do {                                                                    \
        hipError_t err = stmt;                                                \
        if (err != hipSuccess) {                                            \
            cudaLog(ERROR, "Failed to run stmt ", #stmt);                    \
            cudaLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err));    \
            return -1;                                                        \
            }                                                                \
        } while (0)                                                            \


__global__ void histogramKernel(int *vec, int n, unsigned int *d_out) {
    // Taking care of the following: Privatization, using shared memory and using atomic operations, num bins mapping
    // Splitting into sections and determining the number of the sections for each thread
    // and howe much the thread will take

    // Block size is 256 thread, so the elements_per_threads will be 50 in each thread
    // We can make use of the caolesced memories without nothing changed
    __shared__ unsigned int s_bins[NUM_BINS];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculating hist in privatization shared memory
    while (index < n) {
        int count = atomicAdd(&s_bins[vec[index]], 1);
        index += gridDim.x * blockDim.x;
        __syncthreads();

    }

    __syncthreads();

    // Add to the global memory
    if (threadIdx.x == 0)
        for (int j = 0; j < NUM_BINS; j++)
            atomicAdd(&d_out[j], s_bins[j]);

}

__global__ void truncate_output(unsigned int *d_out) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index < NUM_BINS)
        d_out[index] = 127 & d_out[index];
}


__global__ void textHistogramKernel(char *vec, int n, unsigned int *out) {
    // Taking care of the following: Privatization, using shared memory and using atomic operations, num bins mapping
    // Splitting into sections and determining the number of the sections for each thread
    // and howe much the thread will take

    // Block size is 256 thread, so the index_jump will be 50 in each thread
    // We can make use of the caolesced memories without nothing changed
    __shared__ unsigned int s_bins[TEXT_NUM_BINS];
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Initializing the bin counters. This Step can be removed
    if (i < TEXT_NUM_BINS)
        s_bins[i] = 0;
    __syncthreads();

    // Calculating hist in privatization shared memory

    while (i < n) {
        // Atomic add
        atomicAdd(&s_bins[int(vec[i])], 1);
        i += gridDim.x * blockDim.x;
        __syncthreads();
    }

    __syncthreads();

    // Add to the global memory, as the block has 256 threads (more than the number of bins)
    int i_thread = threadIdx.x;
    if (i_thread < TEXT_NUM_BINS) {
        atomicAdd(&out[i_thread], s_bins[i_thread]);
    }

    __syncthreads();

}

void HistogramKernel::runHistogram(int *vec, int n, unsigned int *out) {
    histogramKernel << < this->gridSize, this->blockSize >> > (vec, n, out);
}

void HistogramKernel::runTextHistogram(char *d_in, int n, unsigned int *out) {
    textHistogramKernel << < this->gridSize, this->blockSize >> > (d_in, n, out);
}
#include "hip/hip_runtime.h"
#include "kernel.h"

#define cudaCheck(stmt)                                                        \
    do {                                                                    \
        hipError_t err = stmt;                                                \
        if (err != hipSuccess) {                                            \
            cudaLog(ERROR, "Failed to run stmt ", #stmt);                    \
            cudaLog(ERROR, "Got CUDA error ... ", hipGetErrorString(err));    \
            return -1;                                                        \
            }                                                                \
        } while (0)                                                            \


__global__ void kernel() {
}

void Kernel::run() {
    kernel <<< this->gridSize, this->blockSize >>> ();
}
